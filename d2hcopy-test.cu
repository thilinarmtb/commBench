#include "comm-bench.h"
#include "comm-bench-cuda.h"

inline hipError_t cudaCheck(hipError_t result) {
#if defined(COMMBENCH_DEBUG)
  if(result != hipSuccess) {
    fprintf(stderr,"CUDA runtime error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

void timeCopy(float *h_a, float *d_a, int bytes, int NITER, int rank) {
  hipEvent_t start, stop;
  float time0;
  float time1;

  cudaCheck(hipEventCreate(&start)); 
  cudaCheck(hipEventCreate(&stop)); 

  cudaCheck(hipEventRecord(start, 0));
  cudaCheck(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
  cudaCheck(hipEventRecord(stop, 0));
  cudaCheck(hipEventSynchronize(stop));
  cudaCheck(hipEventElapsedTime(&time0, start, stop));

  cudaCheck(hipEventRecord(start, 0));
  for(int i=0; i<NITER; i++) {
    cudaCheck(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice)); 
  }
  cudaCheck(hipEventRecord(stop, 0));
  cudaCheck(hipEventSynchronize(stop));
  cudaCheck(hipEventElapsedTime(&time1, start, stop));
  time1 /= NITER;

  if(rank == 0) {
    printf("Test=h2d,size=%d,time0=%f,time1=%f\n",bytes,time0, time1);
  }

  cudaCheck(hipEventRecord(start, 0));
  cudaCheck(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost));
  cudaCheck(hipEventRecord(stop, 0));
  cudaCheck(hipEventSynchronize(stop));
  cudaCheck(hipEventElapsedTime(&time0, start, stop));

  cudaCheck(hipEventRecord(start, 0));
  for(int i=0; i<NITER; i++) {
    cudaCheck(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost)); 
  }
  cudaCheck(hipEventRecord(stop, 0));
  cudaCheck(hipEventSynchronize(stop));
  cudaCheck(hipEventElapsedTime(&time1, start, stop));
  time1 /= NITER;

  if(rank == 0) {
    printf("Test=d2h,size=%d,time0=%f,time1=%f\n",bytes,time0, time1);
  }

  cudaCheck(hipEventDestroy(start));
  cudaCheck(hipEventDestroy(stop));

  return;
}

int main(int argc, char **argv){
  MPI_Init(&argc, &argv);
  int rank, np;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &np);

  // > 0 verbose output
  int verbose = 0;

  // get device id based on rank
  int device_id = getDeviceId(MPI_COMM_WORLD, verbose);
  hipSetDevice(device_id);
  if(verbose) printf("device_id at rank=%d is %d\n", rank, device_id);

  int N = atoi(argv[1]);
  int NITER = 100000;
  int size = N*sizeof(float);

  float *ha;
  float *da;

  // Allocate Memory on host
  ha = (float *)malloc(size);

  // Allocate memory on device
  hipMalloc(&da, size);

  // Fill a with bogus values
  for(int i = 0; i < N; i++) {
    ha[i] = 1.0*i + 10.0;
  } 

  if(rank ==0) printf("np = %d\n", np);
  timeCopy(ha, da, size, NITER, rank);

  hipFree(da);
  free(ha);

  MPI_Finalize(); 

  return 0;
}
